#include "hip/hip_runtime.h"
#include "global.h"

#include <math.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <memory>
#include <vector>

using namespace std;

__host__ void OutputPosNormIndex(string filepath, std::vector<float> pos = std::vector<float>(), std::vector<float> norm = std::vector<float>(),
                                            std::vector<unsigned int> index = std::vector<unsigned int>()) {
    ofstream out(filepath);
    if (!out) {
        cout << "OutputPosNormIndex : open file " << filepath << " failed " << endl;
        return;
    }
    for (auto iter = pos.begin(); iter != pos.end();) {
        out << "v " << *(iter) << " ";
        iter++;
        out << *(iter) << " ";
        iter++;
        out << *(iter) << endl;
        iter++;
    }
    for (auto iter = norm.begin(); iter != norm.end();) {
        out << "vn " << *(iter) << " ";
        iter++;
        out << *(iter) << " ";
        iter++;
        out << *(iter) << endl;
        iter++;
    }
    for (auto iter = index.begin(); iter != index.end();) {
        out << "f " << *(iter) + 1 << " ";
        iter++;
        out << *(iter) + 1 << " ";
        iter++;
        out << *(iter) + 1 << endl;
        iter++;
    }
}

__host__ void printCudaError(const char* funcName) {
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "%s error: %s\n", funcName, hipGetErrorString(cudaStatus));
    }
}

__device__ __host__ void MatrixProduct_3_D(const float* A, const float* B, float* R) {
    R[0] = A[0] * B[0] + A[1] * B[3] + A[2] * B[6];
    R[1] = A[0] * B[1] + A[1] * B[4] + A[2] * B[7];
    R[2] = A[0] * B[2] + A[1] * B[5] + A[2] * B[8];
    R[3] = A[3] * B[0] + A[4] * B[3] + A[5] * B[6];
    R[4] = A[3] * B[1] + A[4] * B[4] + A[5] * B[7];
    R[5] = A[3] * B[2] + A[4] * B[5] + A[5] * B[8];
    R[6] = A[6] * B[0] + A[7] * B[3] + A[8] * B[6];
    R[7] = A[6] * B[1] + A[7] * B[4] + A[8] * B[7];
    R[8] = A[6] * B[2] + A[7] * B[5] + A[8] * B[8];
}

__device__ __host__ void MatrixSubstract_3_D(float* A, float* B, float* R) {
    for (int i = 0; i < 9; i++) R[i] = A[i] - B[i];
}

__device__ __host__ void MatrixProduct_D(float* A, float* B, float* R, int nx, int ny, int nz) {
    memset(R, 0, sizeof(float) * nx * nz);
    for (int i = 0; i < nx; i++)
        for (int j = 0; j < nz; j++)
            for (int k = 0; k < ny; k++) R[i * nz + j] += A[i * ny + k] * B[k * nz + j];
}

__device__ __host__ void GetRotation_D(float F[3][3], float R[3][3], float& deltaF) {
    float C[3][3];
    memset(&C[0][0], 0, sizeof(float) * 9);
    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
            for (int k = 0; k < 3; k++) C[i][j] += F[k][i] * F[k][j];

    float C2[3][3];
    memset(&C2[0][0], 0, sizeof(float) * 9);
    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
            for (int k = 0; k < 3; k++) C2[i][j] += C[i][k] * C[j][k];

    float det = F[0][0] * F[1][1] * F[2][2] + F[0][1] * F[1][2] * F[2][0] + F[1][0] * F[2][1] * F[0][2] - F[0][2] * F[1][1] * F[2][0] -
                F[0][1] * F[1][0] * F[2][2] - F[0][0] * F[1][2] * F[2][1];
    deltaF = det;

    float I_c = C[0][0] + C[1][1] + C[2][2];
    float I_c2 = I_c * I_c;
    float II_c = 0.5 * (I_c2 - C2[0][0] - C2[1][1] - C2[2][2]);
    float III_c = det * det;
    float k = I_c2 - 3 * II_c;  // k ��һ��ƽ���ͣ����ڵ��� 0

    float inv_U[3][3];
    if (k < 1e-6f) {                                                       // k == 0
        if (I_c < 1e-6) printf("[ERROR]I_c = %f, �˻���һ���㣿\n", I_c);  // I_c == 0 <=> F = {0}
        float inv_lambda = 1 / sqrt(I_c / 3);
        memset(inv_U, 0, sizeof(float) * 9);
        inv_U[0][0] = inv_lambda;
        inv_U[1][1] = inv_lambda;
        inv_U[2][2] = inv_lambda;
    } else {  // k > 0
        float l = I_c * (I_c * I_c - 4.5 * II_c) + 13.5 * III_c;
        float k_root = sqrt(k);
        float value = l / (k * k_root);
        if (value < -1.0) value = -1.0;
        if (value > 1.0) value = 1.0;
        float phi = acos(value);
        float lambda2 = (I_c + 2 * k_root * cos(phi / 3)) / 3.0;  // phi in [0, pi], phi/3 in [0, pi/3], cos > 0
        float lambda = sqrt(lambda2);

        float III_u = sqrt(III_c);
        if (det < 0) III_u = -III_u;  // ??? �Ի���Ϊ III_u == det

        if (lambda < 1e-6) printf("[ERROR]lambada = %f, Ӧ���Ǵ��� 0 �ģ�\n", lambda);
        if (-lambda2 + I_c + 2 * III_u / lambda < 1e-6)
            printf("[ERROR] -lambda2 + I_c + 2 * III_u / lambda = %f (det = %f)\n", -lambda2 + I_c + 2 * III_u / lambda, det);
        float I_u = lambda + sqrt(-lambda2 + I_c + 2 * III_u / lambda);
        float II_u = (I_u * I_u - I_c) * 0.5;

        float U[3][3];
        float inv_rate, factor;

        if (I_u * II_u - III_u < 1e-6) printf("[ERROR]I_u * II_u - III_u = %f\n", I_u * II_u - III_u);
        inv_rate = 1 / (I_u * II_u - III_u);

        factor = I_u * III_u * inv_rate;

        memset(U, 0, sizeof(float) * 9);
        U[0][0] = factor;
        U[1][1] = factor;
        U[2][2] = factor;

        factor = (I_u * I_u - II_u) * inv_rate;
        for (int i = 0; i < 3; i++)
            for (int j = 0; j < 3; j++) U[i][j] += factor * C[i][j] - inv_rate * C2[i][j];

        if (fabs(III_u) < 1e-6) printf("[ERROR]III_u = %f, det = %f\n", III_u, det);  // ��������Ϊ�������˻���һ��ƽ����
        inv_rate = 1 / III_u;

        factor = II_u * inv_rate;
        memset(inv_U, 0, sizeof(float) * 9);
        inv_U[0][0] = factor;
        inv_U[1][1] = factor;
        inv_U[2][2] = factor;

        factor = -I_u * inv_rate;
        for (int i = 0; i < 3; i++)
            for (int j = 0; j < 3; j++) inv_U[i][j] += factor * U[i][j] + inv_rate * C[i][j];
    }

    memset(&R[0][0], 0, sizeof(float) * 9);
    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
            for (int k = 0; k < 3; k++) R[i][j] += F[i][k] * inv_U[k][j];
}

__device__ __host__ void GetRotation_D(float F[3][3], float R[3][3]) {
    float det;
    GetRotation_D(F, R, det);
}

__device__ __host__ float Matrix_Inverse_3(float* A, float* R) {
    R[0] = A[4] * A[8] - A[7] * A[5];
    R[1] = A[7] * A[2] - A[1] * A[8];
    R[2] = A[1] * A[5] - A[4] * A[2];
    R[3] = A[5] * A[6] - A[3] * A[8];
    R[4] = A[0] * A[8] - A[2] * A[6];
    R[5] = A[2] * A[3] - A[0] * A[5];
    R[6] = A[3] * A[7] - A[4] * A[6];
    R[7] = A[1] * A[6] - A[0] * A[7];
    R[8] = A[0] * A[4] - A[1] * A[3];
    float det = A[0] * R[0] + A[3] * R[1] + A[6] * R[2];
    if (fabs(det) < 1e-6) {
        printf("det is %f, �������˻�\n", det);
        // det = 1e-5;
        exit(0);
    }
    float inv_det = 1 / det;
    for (int i = 0; i < 9; i++) R[i] *= inv_det;
    return det;
}

__device__ __host__ float GetVolumn(const Point3D& A, const Point3D& B, const Point3D& C, const Point3D& D) {
    Point3D AB = B - A;
    Point3D AC = C - A;
    Point3D AD = D - A;
    float Deformation[9] = {AB.x, AC.x, AD.x, AB.y, AC.y, AD.y, AB.z, AC.z, AD.z};
    float R[9];
    R[0] = Deformation[4] * Deformation[8] - Deformation[7] * Deformation[5];
    R[1] = Deformation[7] * Deformation[2] - Deformation[1] * Deformation[8];
    R[2] = Deformation[1] * Deformation[5] - Deformation[4] * Deformation[2];
    R[3] = Deformation[5] * Deformation[6] - Deformation[3] * Deformation[8];
    R[4] = Deformation[0] * Deformation[8] - Deformation[2] * Deformation[6];
    R[5] = Deformation[2] * Deformation[3] - Deformation[0] * Deformation[5];
    R[6] = Deformation[3] * Deformation[7] - Deformation[4] * Deformation[6];
    R[7] = Deformation[1] * Deformation[6] - Deformation[0] * Deformation[7];
    R[8] = Deformation[0] * Deformation[4] - Deformation[1] * Deformation[3];
    float det = Deformation[0] * R[0] + Deformation[3] * R[1] + Deformation[6] * R[2];
    if (fabs(det) < 1e-6) {
        printf("det is %f, �������˻�\n", det);
        exit(0);
    }
    return det / 6.0f;
}

__device__ __host__ Point3D operator-(const Point3D& a, const Point3D& b) { return {a.x - b.x, a.y - b.y, a.z - b.z}; }

__device__ __host__ Point3D operator+(const Point3D& a, const Point3D& b) { return {a.x + b.x, a.y + b.y, a.z + b.z}; }

__device__ __host__ Point3D crossProduct(const Point3D& a, const Point3D& b) { return {a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x}; }

__device__ __host__ Point3D operator/(const Point3D& a, float b) { return {a.x / b, a.y / b, a.z / b}; }

__device__ __host__ Point3D operator*(const Point3D& a, float b) { return {a.x * b, a.y * b, a.z * b}; }

__device__ __host__ float dotProduct(const Point3D& a, const Point3D& b) { return a.x * b.x + a.y * b.y + a.z * b.z; }

__device__ __host__ float vectorLength(const Point3D& v) { return std::sqrt(v.x * v.x + v.y * v.y + v.z * v.z); }

__device__ __host__ bool pointInTet(const float* tetVertPos, const float* tetFaceNormal, const float* point) {
    const Point3D P = {point[0], point[1], point[2]};
    for (int k = 0; k < 4; k++) {
        int id_A = (k + 1) % 4;  // ����ѡȡ��һ����
        const Point3D A = {tetVertPos[id_A * 3 + 0], tetVertPos[id_A * 3 + 1], tetVertPos[id_A * 3 + 2]};
        const Point3D AP = P - A;
        const Point3D normal = {tetFaceNormal[k * 3 + 0], tetFaceNormal[k * 3 + 1], tetFaceNormal[k * 3 + 2]};
        float dot = dotProduct(AP, normal);
        if (dot > 1e-6) {
            return false;
        }
    }
    return true;
}

__device__ __host__ vector<float> barycentricCoordinate(const float* point, const float* tetCenter, const float* tetFaceArea, const float* tetFaceNormal,
                                                        float V) {
    Point3D r = {point[0], point[1], point[2]};
    Point3D rB = {tetCenter[0], tetCenter[1], tetCenter[2]};
    float F0 = tetFaceArea[0];
    float F1 = tetFaceArea[1];
    float F2 = tetFaceArea[2];
    float F3 = tetFaceArea[3];
    Point3D n0 = {tetFaceNormal[3 * 0 + 0], tetFaceNormal[3 * 0 + 1], tetFaceNormal[3 * 0 + 2]};
    Point3D n1 = {tetFaceNormal[3 * 1 + 0], tetFaceNormal[3 * 1 + 1], tetFaceNormal[3 * 1 + 2]};
    Point3D n2 = {tetFaceNormal[3 * 2 + 0], tetFaceNormal[3 * 2 + 1], tetFaceNormal[3 * 2 + 2]};
    Point3D n3 = {tetFaceNormal[3 * 3 + 0], tetFaceNormal[3 * 3 + 1], tetFaceNormal[3 * 3 + 2]};

    float lambda0 = 0.25f - dotProduct(r - rB, n0) * F0 / (3 * V);
    float lambda1 = 0.25f - dotProduct(r - rB, n1) * F1 / (3 * V);
    float lambda2 = 0.25f - dotProduct(r - rB, n2) * F2 / (3 * V);
    float lambda3 = 0.25f - dotProduct(r - rB, n3) * F3 / (3 * V);

    vector<float> ans = {lambda0, lambda1, lambda2, lambda3};
    return ans;
}

__device__ __host__ void barycentricCoordinate(const Point3D& point, const Point3D& tetVertA, const Point3D& tetVertB, const Point3D& tetVertC,
                                               const Point3D& tetVertD, float* weights) {
    float V = GetVolumn(tetVertA, tetVertB, tetVertC, tetVertD);
    Point3D center = (tetVertA + tetVertB + tetVertC + tetVertD) * 0.25f;
    Point3D tetVerts[4] = {tetVertA, tetVertB, tetVertC, tetVertD};
    int pointFacePair[4][4] = {{0, 1, 2, 3}, {1, 2, 3, 0}, {2, 3, 0, 1}, {3, 0, 1, 2}};
    for (int i = 0; i < 4; i++) {
        Point3D p = tetVerts[pointFacePair[i][0]];
        Point3D facePoint0 = tetVerts[pointFacePair[i][1]];
        Point3D facePoint1 = tetVerts[pointFacePair[i][2]];
        Point3D facePoint2 = tetVerts[pointFacePair[i][3]];

        Point3D edge01 = facePoint1 - facePoint0;
        Point3D edge02 = facePoint2 - facePoint0;

        Point3D normal = crossProduct(edge01, edge02);
        Point3D edge0p = p - facePoint0;
        if (dotProduct(normal, edge0p) > 0){ // ǰ�������ж�����������Ƿ�Ϊ 0���˴�Ӧ�ò����н�Сֵ
            normal.x = -normal.x;
            normal.y = -normal.y;
            normal.z = -normal.z;
        }

        float len = vectorLength(normal);
        float area = len * 0.5f;
        normal = normal / len;

        weights[i] = 0.25f - dotProduct(point - center, normal) * area / (3 * V);
    }
}